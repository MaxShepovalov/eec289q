#include "hip/hip_runtime.h"
//graph coloring

//CUDA accelerator approach

//color management is happeing on CPU, while for loops are parralellized on GPU

//KernelNeighbourColor(graph_line, colors, output, Vsize);
//Kernel
#include <stdio.h>
#include <stdlib.h>
#include <set>
#include <sstream>
#include <string>
#include <fstream>
#include <iostream>
#include <cstring>
#include <math.h>

//file parsers from example
void ReadColFile(const char filename[], bool** graph, int* V);
void ReadMMFile(const char filename[], bool** graph, int* V);

////////////////////////////////////////////////////////////////////////////////////

// __device__ int DeviceVertexProcess(bool* graph, int* colors, int Vsize, int V1, int V2){
//     //graph - pointer on graph
//     //colors - pointer on current selected colors
//     //Vsize - amount of vertexes in graph
//     //V1 - main vertex index
//     //V2 - vertex index to compare with
    
//     //filtered color
//     int out = 0;
    
//     //if vertexes are connected
//     if (graph[V1 * V + V2]){
//         out = colors[V2];
//     }
    
//     //retrn result
//     return out;
// }


//Kernel work with pairs of vertexes
__global__ void KernelNeighbourColor(bool* graph, int* colors, bool* output, int V){
    int index = floorf(threadIdx.x/V); // primary vertex index (row of graph)
    int near  = threadIdx.x % V//neighbor vertex index         (col of graph)

    //stage 1. scan neighbour

    //find color for neighbour
    //int color_near_r = DeviceVertexProcess(graph, colors, V, V1, index);
    int color_near_r = 0;
    if (graph[index * V + near]){
        color_near_r = colors[near];
    }

    //write color
    //color_near[index] = color_near_r;
    //__syncthreads();

    //stage 2. mark used colors
    if (color_near_r != 0){
        output[index * V + color_near_r] = true;
    }
}

__global__ void KernelSearchColor(int* colors, int* nearcolors, int V){
    int index = threadIdx.x; //vertex index
    for (int clr = 0; clr < V; clr ++){
        if (!nearcolors[index * V + clr]){
            colors[index] = clr;
            break;
        }
    }
}

// __global__ void GraphKernel(bool* graph, int* color, int V) {
//     int index = blockIdx.x * blockDim.x + threadIdx.x; //thread ID
// //    int stride = blockDim.x * gridDim.x;               //
//     //each thread works with only one vertex

//     //shared memory for final colors
//     extern __shared__ int color_sh[];
//     color_sh[index] = 0;
//     __syncthreads();

//     //decide the color
//     for (int attempt = 0; attempt < V; attempt++) {

//         //scan colors of neighbours
//         bool* near = new bool[V+1];
//         for (int i = 0; i < V; i++) near[i] = false;

//         for (int i = 0; i < V; i++) {
//             if (graph[index * V + i] and i != index) {
//                 //near.insert(color_sh[i]);
//                 near[color_sh[i]] = true;
//             }
//         }

//         //select color
//         for (int color_i = 1; color_i < V; color_i++) {
//             if (!near[color_i]) {
//                 color_sh[index] = color_i;
//                 break;
//             }
//         }

//         //wait for others
//         __syncthreads();
        
//         //check if there is a mistake
//         bool done = true;
//         for (int i = index + 1; i < V; i++) {
//             if (graph[index * V + i] and color_sh[i]==color_sh[index]) {
//                 done = false;
//                 break;
//             }
//         }
//         if (done) {
//             //exit loop
//             break;
//         }
//     }
//
//     //write out result
//     color[index] = color_sh[index];
// }

void GraphColoringGPU(const char filename[], int** color){
    int V;         //number of vertexes
    bool* graph_h; //graph matrix on host
    bool* graph_d; //graph matrix on device
    //int* color_d;  //colors on device

    //read graph file
    if (std::string(filename).find(".col") != std::string::npos)
        ReadColFile(filename, &graph_h, &V);
    else if (std::string(filename).find(".mm") != std::string::npos) 
        ReadMMFile(filename, &graph_h, &V);
    else
        //exit now, if cannot parse the file
        return;

    //allocate list of colors per vector
    hipMallocManaged(color, V * sizeof(int));

    //move graph to device memory
    hipMalloc((bool**)&graph_d, V * V * sizeof(bool));
    hipMemcpy(graph_d, graph_h, V * V * sizeof(bool), hipMemcpyHostToDevice);
    
    //start kernel
    //int nblocks = 1;
    //int nthreads = V;
    //GraphKernel<<<nblocks, nthreads, V * sizeof(bool)>>>(graph_d, color_d, V);
    //GraphKernel<<<nblocks, nthreads, V * sizeof(bool)>>>(graph_d, *color, V);
    for (int vi = 0; vi < V; vi++){
        bool* near_colors;
        hipMallocManaged(&near_colors, V * V * sizeof(bool));
        KernelNeighbourColor<<<1, V*V>>>(graph_d, *color, near_colors, V, vi);

        //find colors

        KernelSearchColor<<<1, V>>>(*color, near_colors, V);
        hipFree(near_colors);
        
        //sync CUDA and CPU
        hipError_t synced = hipDeviceSynchronize();
        if (synced != hipSuccess){
            std::cout << "cuda sync ERROR happened: " << hipGetErrorName(synced) << std::endl;
            exit(synced);
        }
        // else {
        //   std::cout << "cuda sync OK" << std::endl;
        // }
    }

    //counter
    int num_colors = 0;
    bool seen_colors[V+1];
    for (int i = 0; i < V; i++) seen_colors[i] = false;

    std::cout << "Vertex - color" << std::endl;

    for (int i = 0; i < V; i++) {
       if (!seen_colors[(*color)[i]]) {
          seen_colors[(*color)[i]] = true;
          num_colors++;
       }  
    }

    //print result
    for (int i = 0; i < V; i++) {
        std::cout << i << " - color " << (*color)[i] << std::endl;
    }
    std::cout << "Solution has " << num_colors << " colors" << std::endl;
    hipFree(*color);
}

/////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char const **argv)
{
    /* code */
    // std::cout << argc << " items " << std::endl;
    // for (int i = 0; i < argc; i++){
    //     std::cout << i << ": '" << argv[i] << "'" << std::endl;
    // }
    int* color;
    GraphColoringGPU(argv[1], &color);
    return 0;
}

/////////////////////////////////////////////////////////////////////////////////////
// Read MatrixMarket graphs
// Assumes input nodes are numbered starting from 1
void ReadMMFile(const char filename[], bool** graph, int* V) 
{
   std::string line;
   std::ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   // Reading comments
   while (getline(infile, line)) {          
      std::istringstream iss(line);
      if (line.find("%") == std::string::npos)
         break;
   }

   // Reading metadata
   std::istringstream iss(line);
   int num_rows, num_cols, num_edges;
   iss >> num_rows >> num_cols >> num_edges;

   *graph = new bool[num_rows * num_rows];
   memset(*graph, 0, num_rows * num_rows * sizeof(bool));
   *V = num_rows;

   // Reading nodes
   while (getline(infile, line)) {          
      std::istringstream iss(line);
      int node1, node2, weight;
      iss >> node1 >> node2 >> weight;
      
      // Assume node numbering starts at 1
      (*graph)[(node1 - 1) * num_rows + (node2 - 1)] = true;
      (*graph)[(node2 - 1) * num_rows + (node1 - 1)] = true;
   }
   infile.close();
}


// Read DIMACS graphs
// Assumes input nodes are numbered starting from 1
void ReadColFile(const char filename[], bool** graph, int* V) 
{
   std::string line;
   std::ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   int num_rows, num_edges;

   while (getline(infile, line)) {
      std::istringstream iss(line);
      std::string s;
      int node1, node2;
      iss >> s;
      if (s == "p") {
         iss >> s; // read string "edge"
         iss >> num_rows;
         iss >> num_edges;
         *V = num_rows;
         *graph = new bool[num_rows * num_rows];
         memset(*graph, 0, num_rows * num_rows * sizeof(bool));
         continue;
      } else if (s != "e")
         continue;
      
      iss >> node1 >> node2;

      // Assume node numbering starts at 1
      (*graph)[(node1 - 1) * num_rows + (node2 - 1)] = true;
      (*graph)[(node2 - 1) * num_rows + (node1 - 1)] = true;
   }
   infile.close();
}