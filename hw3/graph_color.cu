//graph coloring

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <set>
#include <sstream>
#include <string>
#include <fstream>
#include <iostream>
#include <cstring>
#include <math.h>

//file parsers from example
void ReadColFile(const char filename[], bool** graph, int* V);
void ReadMMFile(const char filename[], bool** graph, int* V);

////////////////////////////////////////////////////////////////////////////////////

__global__ void GraphKernel(bool* graph, int* color, int V) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; //thread ID
//    int stride = blockDim.x * gridDim.x;               //
    //each thread works with only one vertex

    //shared memory for final colors
    extern __shared__ int color_sh[];
    color_sh[index] = 0;
    __syncthreads();

    //decide the color
    for (int attempt = 0; attempt < V; attempt++) {

        //scan colors of neighbours
        bool* near = new bool[V+1];
        for (int i = 0; i < V; i++) near[i] = false;

        for (int i = 0; i < V; i++) {
            if (graph[index * V + i] and i != index) {
                //near.insert(color_sh[i]);
                near[color_sh[i]] = true;
            }
        }

        //select color
        for (int color_i = 1; color_i < V; color_i++) {
            if (!near[color_i]) {
                color_sh[index] = color_i;
                break;
            }
        }

        //wait for others
        __syncthreads();
        
        //check if there is a mistake
        bool done = true;
        for (int i = index + 1; i < V; i++) {
            if (graph[index * V + i] and color_sh[i]==color_sh[index]) {
                done = false;
                break;
            }
        }
        if (done) {
            //exit loop
            break;
        }
    }

    //write out result
    color[index] = color_sh[index];
}

void GraphColoringGPU(const char filename[], int** color){
    int V;         //number of vertexes
    bool* graph_h; //graph matrix on host
    bool* graph_d; //graph matrix on device
    int* color_d;  //colors on device

    //read graph file
    if (std::string(filename).find(".col") != std::string::npos)
        ReadColFile(filename, &graph_h, &V);
    else if (std::string(filename).find(".mm") != std::string::npos) 
        ReadMMFile(filename, &graph_h, &V);
    else
        //exit now, if cannot parse the file
        return;

    //allocate list of colors per vector
    //cudaMallocManaged(color, V * sizeof(int));
    hipMalloc((int**)*color_d, V * sizeof(int));

    //move graph to device memory
    hipMalloc((bool**)&graph_d, V * V * sizeof(bool));
    hipMemcpy(graph_d, graph_h, V * V * sizeof(bool), hipMemcpyHostToDevice);
    
    //start kernel
    int nblocks = 1;
    int nthreads = V;
    GraphKernel<<<nblocks, nthreads, V * sizeof(bool)>>>(graph_d, color_d, V);

    //sync CUDA and CPU
    hipError_t synced = hipDeviceSynchronize();
    if (synced != hipSuccess){
        std::cout << "cuda sync ERROR happened: " << hipGetErrorName(synced) << std::endl;
        exit(synced);
    } else {
        std::cout << "cuda sync OK" << std::endl;
    }

    //move colors to host
    hipMemcpy(*color, color_d, V * sizeof(int), hipMemcpyDeviceToHost);

    //counter from example
    int num_colors = 0;
    bool seen_colors[V+1];
    for (int i = 0; i < V; i++) seen_colors[i] = false;

    for (int i = 0; i < V; i++) {
       if (!seen_colors[(*color)[i]]) {
          seen_colors[(*color)[i]] = true;
          num_colors++;
       }  
    }

    //print result
    for (int i = 0; i < V; i++) {
        std::cout << i << " - color " << (*color)[i] << std::endl;
    }
    std::cout << "Solution has " << num_colors << " colors" << std::endl;
}

/////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char const **argv)
{
    /* code */
    // std::cout << argc << " items " << std::endl;
    // for (int i = 0; i < argc; i++){
    //     std::cout << i << ": '" << argv[i] << "'" << std::endl;
    // }
    int* color;
    GraphColoringGPU(argv[1], &color);
    return 0;
}

/////////////////////////////////////////////////////////////////////////////////////
// Read MatrixMarket graphs
// Assumes input nodes are numbered starting from 1
void ReadMMFile(const char filename[], bool** graph, int* V) 
{
   std::string line;
   std::ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   // Reading comments
   while (getline(infile, line)) {          
      std::istringstream iss(line);
      if (line.find("%") == std::string::npos)
         break;
   }

   // Reading metadata
   std::istringstream iss(line);
   int num_rows, num_cols, num_edges;
   iss >> num_rows >> num_cols >> num_edges;

   *graph = new bool[num_rows * num_rows];
   memset(*graph, 0, num_rows * num_rows * sizeof(bool));
   *V = num_rows;

   // Reading nodes
   while (getline(infile, line)) {          
      std::istringstream iss(line);
      int node1, node2, weight;
      iss >> node1 >> node2 >> weight;
      
      // Assume node numbering starts at 1
      (*graph)[(node1 - 1) * num_rows + (node2 - 1)] = true;
      (*graph)[(node2 - 1) * num_rows + (node1 - 1)] = true;
   }
   infile.close();
}


// Read DIMACS graphs
// Assumes input nodes are numbered starting from 1
void ReadColFile(const char filename[], bool** graph, int* V) 
{
   std::string line;
   std::ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   int num_rows, num_edges;

   while (getline(infile, line)) {
      std::istringstream iss(line);
      std::string s;
      int node1, node2;
      iss >> s;
      if (s == "p") {
         iss >> s; // read string "edge"
         iss >> num_rows;
         iss >> num_edges;
         *V = num_rows;
         *graph = new bool[num_rows * num_rows];
         memset(*graph, 0, num_rows * num_rows * sizeof(bool));
         continue;
      } else if (s != "e")
         continue;
      
      iss >> node1 >> node2;

      // Assume node numbering starts at 1
      (*graph)[(node1 - 1) * num_rows + (node2 - 1)] = true;
      (*graph)[(node2 - 1) * num_rows + (node1 - 1)] = true;
   }
   infile.close();
}