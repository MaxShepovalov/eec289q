//graph coloring

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <set>
#include <sstream>
#include <string>
#include <fstream>
#include <iostream>
#include <cstring>
#include <math.h>

//file parcers from example
void ReadColFile(const char filename[], bool** graph, int* V);
void ReadMMFile(const char filename[], bool** graph, int* V);

__global__ void KroneckerKernelSmall(int M, int N, float* A, float* B, float* C);

void GraphColoringGPU(const char filename[], int** color);

int main(int argc, char const *argv[])
{
    /* code */
    std::cout << argc << " items " << std::endl;
    for (int i = 0; i < argc; i++){
        std::cout << i << ": '" << argv[argc] << "'" << std::endl;
    }
    return 0;
}

/////////////////////////////////////////////////////////////////////////////////////
// Read MatrixMarket graphs
// Assumes input nodes are numbered starting from 1
void ReadMMFile(const char filename[], bool** graph, int* V) 
{
   std::string line;
   std::ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   // Reading comments
   while (getline(infile, line)) {          
      std::istringstream iss(line);
      if (line.find("%") == std::string::npos)
         break;
   }

   // Reading metadata
   std::istringstream iss(line);
   int num_rows, num_cols, num_edges;
   iss >> num_rows >> num_cols >> num_edges;

   *graph = new bool[num_rows * num_rows];
   memset(*graph, 0, num_rows * num_rows * sizeof(bool));
   *V = num_rows;

   // Reading nodes
   while (getline(infile, line)) {          
      std::istringstream iss(line);
      int node1, node2, weight;
      iss >> node1 >> node2 >> weight;
      
      // Assume node numbering starts at 1
      (*graph)[(node1 - 1) * num_rows + (node2 - 1)] = true;
      (*graph)[(node2 - 1) * num_rows + (node1 - 1)] = true;
   }
   infile.close();
}


// Read DIMACS graphs
// Assumes input nodes are numbered starting from 1
void ReadColFile(const char filename[], bool** graph, int* V) 
{
   std::string line;
   std::ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   int num_rows, num_edges;

   while (getline(infile, line)) {
      std::istringstream iss(line);
      std::string s;
      int node1, node2;
      iss >> s;
      if (s == "p") {
         iss >> s; // read string "edge"
         iss >> num_rows;
         iss >> num_edges;
         *V = num_rows;
         *graph = new bool[num_rows * num_rows];
         memset(*graph, 0, num_rows * num_rows * sizeof(bool));
         continue;
      } else if (s != "e")
         continue;
      
      iss >> node1 >> node2;

      // Assume node numbering starts at 1
      (*graph)[(node1 - 1) * num_rows + (node2 - 1)] = true;
      (*graph)[(node2 - 1) * num_rows + (node1 - 1)] = true;
   }
   infile.close();
}