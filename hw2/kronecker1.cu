#include "hip/hip_runtime.h"
#include "stdio.h"
#include <iostream>
#include <math.h>

//optimized kernel for small matrix B
__global__ void KroneckerKernelSmall(int M, int N, float* A, float* B, float* C){

  //find position
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  //optimize matrix B (load whole array B to SM)
  extern __shared__ float vs[];
  for (int i = 0; i < N*N; i += blockDim.x) {

    //find index of B to copy from
    int b_ind = i + threadIdx.x;
    if ( b_ind < N*N){
      vs[b_ind] = B[b_ind];
//3 debug lines
//      if (blockIdx.x == 252 and (b_ind == 511 or b_ind == 512)){
//        printf("blk:%d, thr:%d Loads B[%d] check:%f expect:%f\n",blockIdx.x,threadIdx.x,index,vs[b_ind],B[b_ind]);
//      }
    }
  }

  //sync all threads
  __syncthreads();

  //row and column calculation for output from current index
  int row = floorf(index/(M*N));
  int col = index%(M*N);

  //A and B index calc
  int a_idx = floorf(row/N) * M + floorf(col/N);
  int b_idx = (row%N) * N + (col%N);

  //actual compute
  C[index] = A[a_idx] * vs[b_idx];

//4 debug lines
//  if (row==8 and (col==7 or col==8)){
//  if (blockIdx.x == 1 and threadIdx.x == 1){
//    printf("blk:%d, thr:%d, bDIM:%d, gDIM:%d, Performs r:%d c:%d C[%d]=A[%d]*B[%d], C = %f, A=%f, B=%f\n",blockIdx.x,threadIdx.x,blockDim.x,gridDim.x,row,col,index,a_idx,b_idx,C[index],A[a_idx],vs[b_idx]);
//  }
}

//optimized code for small matrix B
void KroneckerGPUSmall(int M, int N, float* A, float* B, float* C){

  //number of threads (512 maximum)
  int nthreads = min(N*M*N*M, 512); //512 maximum

  //number of blocks (grid size), find value to match the given size of data
  int nblocks = ceil(N*M*N*M/nthreads);

  //launch the kernel
  //                     nblocks, nthreads, SM size
  KroneckerKernelSmall<<<nblocks, nthreads, N*N*sizeof(float)>>>(M,N,A,B,C);
}

//from reference.cpp
void KroneckerCPU(int M, int N, float* A, float* B, float* C){

  for (int rowA = 0; rowA < M; rowA++){

    for (int colA = 0; colA < M; colA++){
      float elemA = A[rowA * M + colA];

      for (int rowB = 0; rowB < N; rowB++){
        int rowC = rowA * N + rowB;

        for (int colB = 0; colB < N; colB++){
          int colC = colA * N + colB;
          float elemB = B[rowB * N + colB];
//debug line
//          std::cout << "Processing C[" << rowC << "," << colC << "] with A[" << rowA << "," << colA << "] and B[" << rowB << "," << colB << "]" << std::endl;
          C[rowC * (M * N) + colC] = elemA * elemB;
        }
      }
    }
  }
}

//main
int main(){
  int N,M;
  float *A, *B, *C;

  //set matrices dimentions (<16kB, N < 64) (M+N):
  M = 256; //size of A
  N = 64; //size of B

  hipMallocManaged(&A, M*M*sizeof(float));
  hipMallocManaged(&B, N*N*sizeof(float));
  hipError_t malC = hipMallocManaged(&C, N*N*M*M*sizeof(float));
  float* Ccpu = (float*) malloc(sizeof(float) * M * N * M * N);

  if (malC != hipSuccess) {
    std::cout << "Cannot allocate C, err: " << hipGetErrorName(malC) << std::endl;
    exit(malC);
  }

  //fill arrays
  for (int i=0; i < M*M; i++){
    A[i] = i+1;
  }
  for (int i=0; i < N*N; i++){
    B[i] = i+1;
  }
  for (int i=0; i < N*N*M*M; i++){
    C[i] = 0.0f;
    Ccpu[i] = 0.0f;
  }

  //annonce
  std::cout << "Small B optimized computation A:" << M  << " B:" << N << " C:" << N*M << std::endl;

  //compute reference
  std::cout << "CPU start" << std::endl;
  KroneckerCPU(M,N,A,B,Ccpu);
  std::cout << "CPU end" << std::endl;

  //compute answer
  std::cout << "GPU start" << std::endl;
  KroneckerGPUSmall(M,N,A,B,C);
  //KroneckerGPU(M,N,A,B,C);

  //sync CUDA and CPU
  hipError_t synced = hipDeviceSynchronize();
  if (synced != hipSuccess){
    std::cout << "cuda sync ERROR happened " << hipGetErrorName(synced) << std::endl;
    exit(synced);
  } else {
    std::cout << "cuda sync OK" << std::endl;
  }

  std::cout << "GPU end" << std::endl;

  //print mismatches
  std::cout << "looking for mismatches" << std::endl;
  int miss = 0;
  int match = 0;
  for (int row=0; row<M*N; row++){
    for (int columns=0; columns<N*M; columns++){
      int i = row * N * M + columns;
      if (fabs(C[i]-Ccpu[i]) > 0.01) {   //Bus error
        miss++;
        if (miss < 10) {
          std::cout << "row " << row << ", col " << columns;
          std::cout << " Mismatch: GPU " << C[i] << ", CPU " << Ccpu[i] << std::endl;
        } else if (miss == 10) {
          std::cout << "and more ..." << std::endl;
        }
      }
      else {
        match++;
//2 debug lines
//        std::cout << "row " << row << ", col " << columns;
//        std::cout << " OK" << std::endl;
      }
    }
  }
  std::cout << "Found " << miss << " mismatches" << std::endl;
  std::cout << "      " << match << " OK" << std::endl;

//  std::cout << "Search done" << std::endl << "Free pointers" << std::endl;
  hipFree(A);
  hipFree(B);
  hipFree(C);
  free(Ccpu);
  return 0;
}
